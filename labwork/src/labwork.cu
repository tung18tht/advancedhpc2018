#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    switch (lwNum) {
        case 1:
            timer.start();
            labwork.labwork1_CPU();
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork1-cpu-out.jpg");
            timer.start();
            labwork.labwork1_OpenMP();
            printf("labwork 1 OpenMP ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork1-openmp-out.jpg");
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] +
                                          (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));

    #pragma omp parallel for

    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {

}

void Labwork::labwork3_GPU() {

}

void Labwork::labwork4_GPU() {

}

// CPU implementation of Gaussian Blur
void Labwork::labwork5_CPU() {
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,
                     0, 3, 13, 22, 13, 3, 0,
                     1, 13, 59, 97, 59, 13, 1,
                     2, 22, 97, 159, 97, 22, 2,
                     1, 13, 59, 97, 59, 13, 1,
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
    for (int row = 0; row < inputImage->height; row++) {
        for (int col = 0; col < inputImage->width; col++) {
            int sum = 0;
            int c = 0;
            for (int y = -3; y <= 3; y++) {
                for (int x = -3; x <= 3; x++) {
                    int i = col + x;
                    int j = row + y;
                    if (i < 0) continue;
                    if (i >= inputImage->width) continue;
                    if (j < 0) continue;
                    if (j >= inputImage->height) continue;
                    int tid = j * inputImage->width + i;
                    unsigned char gray = (inputImage->buffer[tid * 3] + inputImage->buffer[tid * 3 + 1] + inputImage->buffer[tid * 3 + 2])/3;
                    int coefficient = kernel[(y+3) * 7 + x + 3];
                    sum = sum + gray * coefficient;
                    c += coefficient;
                }
            }
            sum /= c;
            int posOut = row * inputImage->width + col;
            outputImage[posOut * 3] = outputImage[posOut * 3 + 1] = outputImage[posOut * 3 + 2] = sum;
        }
    }
}

void Labwork::labwork5_GPU() {

}

void Labwork::labwork6_GPU() {

}

void Labwork::labwork7_GPU() {

}

void Labwork::labwork8_GPU() {

}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}
