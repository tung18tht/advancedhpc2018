#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    switch (lwNum) {
        case 1:
            timer.start();
            labwork.labwork1_CPU();
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork1-cpu-out.jpg");
            timer.start();
            labwork.labwork1_OpenMP();
            printf("labwork 1 OpenMP ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork1-openmp-out.jpg");
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            timer.start();
            labwork.labwork3_GPU();
            printf("labwork 3 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            timer.start();
            labwork.labwork4_GPU();
            printf("labwork 4 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            timer.start();
            labwork.labwork5_CPU();
            printf("labwork 5 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            timer.start();
            labwork.labwork5_GPU();
            printf("labwork 5 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            timer.start();
            labwork.labwork6_GPU();
            printf("labwork 6 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            timer.start();
            labwork.labwork7_GPU();
            printf("labwork 7 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            timer.start();
            labwork.labwork8_GPU();
            printf("labwork 8 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            timer.start();
            labwork.labwork9_GPU();
            printf("labwork 9 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            timer.start();
            labwork.labwork10_GPU();
            printf("labwork 10 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] +
                                          (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));

    #pragma omp parallel for

    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int noOfGPUs = 0;
    hipGetDeviceCount(&noOfGPUs);

    for (int i = 0; i < noOfGPUs; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        printf("\nGPU #%d:\n", i);
        printf(" - Name: %s\n", prop.name);
        printf(" - Core info:\n");
        printf("    + Clock rate: %d\n", prop.clockRate);
        printf("    + Number of cores: %d\n", getSPcores(prop));
        printf("    + Number of multiprocessors: %d\n", prop.multiProcessorCount);
        printf("    + Warp size: %d\n", prop.warpSize);
        printf(" - Memory info:\n");
        printf("    + Clock rate: %d\n", prop.memoryClockRate);
        printf("    + Bus width: %d\n", prop.memoryBusWidth);
        printf("    + Bandwidth: %d\n", prop.memoryClockRate * prop.memoryBusWidth);
    }
}

__global__ void grayscale(char *input, char *output) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    output[tid * 3] = (input[tid * 3] + input[tid * 3 + 1] + input[tid * 3 + 2]) / 3;
    output[tid * 3 + 2] = output[tid * 3 + 1] = output[tid * 3];
}

void Labwork::labwork3_GPU() {
    int pixelCount = inputImage->width * inputImage->height;

    outputImage = (char *) malloc(pixelCount * 3);
    char* devInput;
    char* devOutput;

    hipMalloc(&devInput, pixelCount * 3);
    hipMalloc(&devOutput, pixelCount * 3);

    hipMemcpy(devInput, inputImage->buffer, pixelCount * 3, hipMemcpyHostToDevice);

    int blockSize = 1024;
    int numBlock = pixelCount / blockSize;

    for (int i = 0; i < 100; ++i)
    {
        grayscale<<<numBlock, blockSize>>>(devInput, devOutput);
    }

    hipMemcpy(outputImage, devOutput, pixelCount * 3, hipMemcpyDeviceToHost);

    hipFree(devInput);
    hipFree(devOutput);
}

__global__ void grayscale2D(char *input, char *output) {
    int globalIdX = threadIdx.x + blockIdx.x * blockDim.x;
    int globalIdY = threadIdx.y + blockIdx.y * blockDim.y;
    int globalId = globalIdX + globalIdY * gridDim.x * blockDim.x;

    // int globalBlockIdx = blockIdx.x + gridDim.x * blockIdx.y;
    // int globalId = globalBlockIdx * blockDim.x * blockDim.y + (threadIdx.x + blockDim.x * threadIdx.y);

    output[globalId * 3] = (input[globalId * 3] + input[globalId * 3 + 1] + input[globalId * 3 + 2]) / 3;
    output[globalId * 3 + 2] = output[globalId * 3 + 1] = output[globalId * 3];
}

void Labwork::labwork4_GPU() {
    int pixelCount = inputImage->width * inputImage->height;

    outputImage = (char *) malloc(pixelCount * 3);
    char* devInput;
    char* devOutput;

    hipMalloc(&devInput, pixelCount * 3);
    hipMalloc(&devOutput, pixelCount * 3);

    hipMemcpy(devInput, inputImage->buffer, pixelCount * 3, hipMemcpyHostToDevice);

    dim3 blockSize = dim3(32, 32);
    dim3 gridSize = dim3(inputImage->width / 32 + 1, inputImage->height / 32 + 1);

    for (int i = 0; i < 100; ++i)
    {
        grayscale2D<<<gridSize, blockSize>>>(devInput, devOutput);
    }

    hipMemcpy(outputImage, devOutput, pixelCount * 3, hipMemcpyDeviceToHost);

    hipFree(devInput);
    hipFree(devOutput);
}

// CPU implementation of Gaussian Blur
void Labwork::labwork5_CPU() {
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,
                     0, 3, 13, 22, 13, 3, 0,
                     1, 13, 59, 97, 59, 13, 1,
                     2, 22, 97, 159, 97, 22, 2,
                     1, 13, 59, 97, 59, 13, 1,
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
    for (int row = 0; row < inputImage->height; row++) {
        for (int col = 0; col < inputImage->width; col++) {
            int sum = 0;
            int c = 0;
            for (int y = -3; y <= 3; y++) {
                for (int x = -3; x <= 3; x++) {
                    int i = col + x;
                    int j = row + y;
                    if (i < 0) continue;
                    if (i >= inputImage->width) continue;
                    if (j < 0) continue;
                    if (j >= inputImage->height) continue;
                    int tid = j * inputImage->width + i;
                    unsigned char gray = (inputImage->buffer[tid * 3] + inputImage->buffer[tid * 3 + 1] + inputImage->buffer[tid * 3 + 2])/3;
                    int coefficient = kernel[(y+3) * 7 + x + 3];
                    sum = sum + gray * coefficient;
                    c += coefficient;
                }
            }
            sum /= c;
            int posOut = row * inputImage->width + col;
            outputImage[posOut * 3] = outputImage[posOut * 3 + 1] = outputImage[posOut * 3 + 2] = sum;
        }
    }
}

void Labwork::labwork5_GPU() {

}

void Labwork::labwork6_GPU() {

}

void Labwork::labwork7_GPU() {

}

void Labwork::labwork8_GPU() {

}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}
